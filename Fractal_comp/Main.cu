#include <stdio.h>
#include "opencv2/highgui/highgui.hpp"
#include <iostream>
#include <conio.h>
#include <fstream>
#include <malloc.h>
#include <assert.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "fenc.h"
#include "fdec.h"

#define IMAGE "lena256.BMP"
#define FILE "lena256.frct"

using namespace std;
using namespace cv;

int main()
{
	hipError_t cudaStatus;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) 
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		_getch();
		return -1;
	}
	
	Mat img = imread(IMAGE, CV_LOAD_IMAGE_UNCHANGED);
	if (img.data == NULL)
	{
		cout << "Image cannot be loaded..!!" << endl;
		system("pause");
		return -1;
	}

	int sv, sh;
	sv = img.rows;
	sh = img.cols;
	if (sv != sh)
	{
		cout << "\nImage is not square";
		return 1;
	}

	int count = 0;
	int *r, *g, *b, *Tr, *Tb, *Tg;
	r = (int*)malloc(img.total()*sizeof(int));
	b = (int*)malloc(img.total()*sizeof(int));
	g = (int*)malloc(img.total()*sizeof(int));
	int k = 0;
	for (int i = 0; i<img.rows; i++) {
		for (int j = 0; j<img.cols; j++) {
			Vec3b p = img.at<Vec3b>(i, j);
			r[k] = int(p[2]);
			g[k] = int(p[1]);
			b[k] = int(p[0]);
			k++;

		}
	}

	int rsize = 4;
	int nd, nr;
	nd = (sv / rsize) / 2;
	nr = sv / rsize;

	Tr = (int*)malloc(nr*nr * 5 * sizeof(int));
	Tb = (int*)malloc(nr*nr * 5 * sizeof(int));
	Tg = (int*)malloc(nr*nr * 5 * sizeof(int));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipDeviceSynchronize();
	hipEventRecord(start, 0);

	cudaStatus = cuda_encoder(r, b, g, Tr, Tb, Tg, rsize, nd, nr, sv, sh);

	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, stop, start);

	cout << "\nEncoding time: " << elapsedTime << " milliseconds\n";

	ofstream fout;
	fout.open(FILE);
	k = nr*nr * 5;
	for (int i = 0; i < k; i++)
		fout << Tr[i] << " "; //writing ith character of array in the file
	fout << "\n";
	for (int i = 0; i < k; i++)
		fout << Tg[i] << " "; //writing ith character of array in the file
	fout << "\n";
	for (int i = 0; i < k; i++)
		fout << Tb[i] << " "; //writing ith character of array in the file
	fout << "\n";

	fout.close();

	int *r1, *g1, *b1, *Tr1, *Tb1, *Tg1;

	Tr1 = (int*)malloc(nr*nr * 5 * sizeof(int));
	Tb1 = (int*)malloc(nr*nr * 5 * sizeof(int));
	Tg1 = (int*)malloc(nr*nr * 5 * sizeof(int));
	r1 = (int*)malloc(img.total()*sizeof(int));
	b1 = (int*)malloc(img.total()*sizeof(int));
	g1 = (int*)malloc(img.total()*sizeof(int));

	ifstream fin;
	fin.open(FILE);
	string line;
	int value;

	k = 0;
	if (getline(fin, line))
	{
		std::istringstream iss(line);
		while (iss >> value)
		{
			Tr1[k] = value;
			k++;
		}
	}
	k = 0;
	if (getline(fin, line))
	{
		std::istringstream iss(line);
		while (iss >> value)
		{
			Tg1[k] = value;
			k++;
		}
	}
	k = 0;
	if (getline(fin, line))
	{
		std::istringstream iss(line);
		while (iss >> value)
		{
			Tb1[k] = value;
			k++;
		}
	}

	hipDeviceSynchronize();
	hipEventRecord(start, 0);

	cudaStatus = cuda_decoder(r1, b1, g1, Tr1, Tb1, Tg1, rsize, nd, nr, sv, sh);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);

	cout << "\nDecoding time: " << elapsedTime << " milliseconds\n";

	Mat A(img.rows, img.cols, CV_8UC3, Scalar(0, 0, 0));

	k = 0;
	for (int i = 0; i<A.rows; i++){
		for (int j = 0; j<A.cols; j++){
			A.data[A.channels()*(A.cols*i + j) + 2] = r1[k]; 
			A.data[A.channels()*(A.cols*i + j) + 1] = g1[k]; 
			A.data[A.channels()*(A.cols*i + j) + 0] = b1[k];
			k++;
		}
	}

	namedWindow("MyImage", CV_WINDOW_AUTOSIZE);

	imshow("MyImage", A);

	

    // Add vectors in parallel.
    /*hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }*/


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	waitKey(0);
	free(r1);
	free(b1);
	free(g1);
	free(Tr1);
	free(Tb1);
	free(Tg1);
	free(r);
	free(b);
	free(g);
	free(Tr);
	free(Tb);
	free(Tg);

	return 0;
}