#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "opencv2/highgui/highgui.hpp"
#include <iostream>
#include <conio.h>
#include <fstream>
#include <malloc.h>
#include <assert.h>

#include "fenc.h"
#include "fdec.h"

#define IMAGE "lena256.BMP"
#define FILE "lena256.frct"

using namespace std;
using namespace cv;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	Mat img = imread(IMAGE, CV_LOAD_IMAGE_UNCHANGED);
	if (img.data == NULL)
	{
		cout << "Image cannot be loaded..!!" << endl;
		system("pause");
		return -1;
	}

	int sv, sh;
	sv = img.rows;
	sh = img.cols;
	if (sv != sh)
	{
		cout << "\nImage is not square";
		return 1;
	}

	int count = 0;
	int *r, *g, *b, *Tr, *Tb, *Tg;
	r = (int*)malloc(img.total()*sizeof(int));
	b = (int*)malloc(img.total()*sizeof(int));
	g = (int*)malloc(img.total()*sizeof(int));
	int k = 0;
	for (int i = 0; i<img.rows; i++) {
		for (int j = 0; j<img.cols; j++) {
			Vec3b p = img.at<Vec3b>(i, j);
			r[k] = int(p[2]);
			g[k] = int(p[1]);
			b[k] = int(p[0]);
			k++;

		}
	}

	int rsize = 4;
	int nd, nr;
	nd = (sv / rsize) / 2;
	nr = sv / rsize;

	Tr = (int*)malloc(nr*nr * 5 * sizeof(int));
	Tb = (int*)malloc(nr*nr * 5 * sizeof(int));
	Tg = (int*)malloc(nr*nr * 5 * sizeof(int));

	fenc(r, Tr, rsize, nd, nr, sv, sh);
	fenc(b, Tb, rsize, nd, nr, sv, sh);
	fenc(g, Tg, rsize, nd, nr, sv, sh);

	ofstream fout;
	fout.open(FILE);
	k = nr*nr * 5;
	for (int i = 0; i < k; i++)
		fout << Tr[i] << " "; //writing ith character of array in the file
	fout << "\n";
	for (int i = 0; i < k; i++)
		fout << Tb[i] << " "; //writing ith character of array in the file
	fout << "\n";
	for (int i = 0; i < k; i++)
		fout << Tg[i] << " "; //writing ith character of array in the file
	fout << "\n";

	fout.close();

	int *r1, *g1, *b1, *Tr1, *Tb1, *Tg1;

	Tr1 = (int*)malloc(nr*nr * 5 * sizeof(int));
	Tb1 = (int*)malloc(nr*nr * 5 * sizeof(int));
	Tg1 = (int*)malloc(nr*nr * 5 * sizeof(int));
	r1 = (int*)malloc(img.total()*sizeof(int));
	b1 = (int*)malloc(img.total()*sizeof(int));
	g1 = (int*)malloc(img.total()*sizeof(int));

	ifstream fin;
	fin.open(FILE);
	string line;
	int value;

	k = 0;
	if (getline(fin, line))
	{
		std::istringstream iss(line);
		while (iss >> value)
		{
			Tr1[k] = value;
			k++;
		}
	}
	k = 0;
	if (getline(fin, line))
	{
		std::istringstream iss(line);
		while (iss >> value)
		{
			Tb1[k] = value;
			k++;
		}
	}
	k = 0;
	if (getline(fin, line))
	{
		std::istringstream iss(line);
		while (iss >> value)
		{
			Tg1[k] = value;
			k++;
		}
	}

	fdec(r1, Tr1, rsize, nd, nr, sv, sh);
	fdec(b1, Tb1, rsize, nd, nr, sv, sh);
	fdec(g1, Tg1, rsize, nd, nr, sv, sh);

	Mat A(img.rows, img.cols, CV_8UC3, Scalar(0, 0, 0));

	k = 0;
	for (int i = 0; i<A.rows; i++){
		for (int j = 0; j<A.cols; j++){
			A.data[A.channels()*(A.cols*i + j) + 0] = b1[k];
			A.data[A.channels()*(A.cols*i + j) + 1] = g1[k];
			A.data[A.channels()*(A.cols*i + j) + 2] = r1[k];
			k++;
		}
	}

	namedWindow("MyImage", CV_WINDOW_AUTOSIZE);

	imshow("MyImage", A);

	

    // Add vectors in parallel.
    /*hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }*/


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	waitKey(0);
	free(r1);
	free(b1);
	free(g1);
	free(Tr1);
	free(Tb1);
	free(Tg1);
	free(r);
	free(b);
	free(g);
	free(Tr);
	free(Tb);
	free(Tg);

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
