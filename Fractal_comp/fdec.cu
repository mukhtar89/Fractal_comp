#include <iostream>
#include <stdio.h>
#include <conio.h>
#include <math.h>
#include <assert.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "mutil.h"

using namespace std;

hipError_t cuda_decoder(int *h_r, int *h_g, int *h_b, int *h_Tr, int *h_Tg, int *h_Tb, int rsize, int nd, int nr, int sv, int sh)
{

	int *d_r, *d_g, *d_b, *d_Tr, *d_Tg, *d_Tb, *d_M1_r, *d_M1_g, *d_M1_b, *d_bigM_r, *d_bigM_g, *d_bigM_b;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&d_r, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_g, sv * sh*sizeof(int));
	cudaStatus = hipMalloc((void**)&d_b, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_Tr, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_Tg, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_Tb, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_M1_r, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_M1_g, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_M1_b, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_bigM_r, nd*nd*rsize*rsize * 8 * sizeof(int));
	cudaStatus = hipMalloc((void**)&d_bigM_g, nd*nd*rsize*rsize * 8 * sizeof(int));
	cudaStatus = hipMalloc((void**)&d_bigM_b, nd*nd*rsize*rsize * 8 * sizeof(int));

	cudaStatus = hipMemcpy(d_r, h_r, sv * sh* sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_g, h_g, sv * sh* sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_b, h_b, sv * sh* sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_Tr, h_Tr, sv * sh* sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_Tg, h_Tg, sv * sh* sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_Tb, h_Tb, sv * sh* sizeof(int), hipMemcpyHostToDevice);

	//kernel call here

	cudaStatus = hipMemcpy(h_Tr, d_Tr, sv * sh* sizeof(int), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(h_Tg, d_Tg, sv * sh* sizeof(int), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(h_Tb, d_Tb, sv * sh* sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	return cudaStatus;
}