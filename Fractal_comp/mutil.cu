#include <iostream>
#include <stdio.h>
#include <conio.h>
#include <math.h>
#include <assert.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "mutil.h"

__device__ void maccess(int *A, int *R, int a1, int a2, int b1, int b2, int size)
{
	for (int a = a1; a <= a2; a++)
		for (int b = b1; b <= b2; b++)
			R[((a - a1)*(b2 - b1 + 1) + (b - b1))] = A[(a*size + b)];
}

__device__ void maccess(int *A, int *R, int a1, int a2, int b1, int b2, int c, int depth, int size)
{
	for (int a = a1; a <= a2; a++)
		for (int b = b1; b <= b2; b++)
			R[((a - a1)*(b2 - b1 + 1) + (b - b1))] = A[((a*size + b)*depth + c)];
}

__device__ void msave(int *A, int *R, int a1, int a2, int b1, int b2, int c, int depth, int size)
{
	for (int a = a1; a <= a2; a++)
		for (int b = b1; b <= b2; b++)
			R[((a*size + b)*depth + c)] = A[((a - a1)*(b2 - b1 + 1) + (b - b1))];
}

__device__  void msave(int *A, int *R, int a1, int a2, int b1, int b2, int size)
{

	for (int a = a1; a <= a2; a++)
		for (int b = b1; b <= b2; b++)
			R[(a*size + b)] = A[((a - a1)*(b2 - b1 + 1) + (b - b1))];
}

__device__ int mavg(int *R, int size)
{
	int sum = msum(R, size);
	int avg = sum / size;
	return avg;
}

__device__ void diff(int *A, int *B, int *C, int size) //each element of A-B = C
{
	for (int i = 0; i < size; i++)
		C[i] = A[i] - B[i];
}

__device__ void msquare(int *A, int *B, int size) //B = A.^2
{
	for (int i = 0; i < size; i++)
		B[i] = A[i] * A[i];
}

__device__ int msum(int *A, int size) //Sum of all elements of A
{
	int sum = 0;
	for (int i = 0; i < size; i++)
		sum += A[i];
	return sum;
}

__device__ void ones(int *A, int k, int size)   //initialize Array A with scalar 'k'
{
	for (int i = 0; i < size; i++)
		A[i] = k;
}

__device__ void matsum(int *A, int *B, int *C, int size)  // add array A + B = C
{
	for (int i = 0; i < size; i++)
		C[i] = A[i] + B[i];
}

__device__ void rotmat(int *D, int *rot, int size)
{
	for (int i = 0; i <size; i++)
		for (int j = 0; j < size; j++)
			rot[i*size + j] = D[(size - j - 1)*size + i];
}

__device__ void transpose(int *D, int *rot, int size)
{
	for (int i = 0; i < size; i++)
		for (int j = i; j < size; j++)
			rot[i*size + j] = D[j*size + i];
}

__device__ void flipv(int *D, int *rot, int size)
{
	for (int i = 0; i < size; i++)
		for (int j = 0; j < size; j++)
			rot[i*size + j] = D[i*size + size - j - 1];
}

__device__ void fliph(int *D, int *rot, int size)
{
	for (int i = 0; i < size; i++)
		for (int j = 0; j < size; j++)
			rot[i*size + j] = D[(size - i - 1)*size + j];
}

__device__ void scale(int *D, float k, int size)
{
	for (int i = 0; i < size; i++)
		D[i] = D[i] * k;
}

__device__ void increment(int *D, float k, int size)
{
	for (int i = 0; i < size; i++)
		D[i] = D[i] + k;
}

__device__ void matcpy(int *A, int *B, int size)
{
	for (int i = 0; i < size; i++)
		A[i] = B[i];
}