#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <conio.h>
#include <math.h>
#include <assert.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include "mutil.h"

#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

using namespace std;

__global__ void cuda_M1(int *M, int *M1, int rsize, int nd, int sv)
{
	__shared__ int *temp;
	__shared__ int stride;
	__shared__ int block;

	temp = new int[4];
	block = 1;
	stride = (rsize*nd) / (1024 * blockDim.x);
	int j = threadIdx.x % (rsize*nd);
	int k = threadIdx.x / (rsize*nd);

	for (int i = k; i < (rsize*nd); i += stride)
	{
		maccess (M, temp, i * 2, i * 2 + 1, j * 2, j * 2 + 1, sv);
		M1[((i*rsize*nd) + j)] = mavg(temp, 4);
	}

	delete[] temp;
}

__global__  void cuda_bigM(int *d_M1, int *d_bigM, int rsize, int nd, int nr, int sv, int sh)
{
	__shared__ int stride;
	stride = (rsize*nd) / (1024 * blockDim.x);
	__shared__ int *D, *temp, *temp2;
	__shared__ int i1, i2, j1, j2;
	D = new int[rsize];
	temp = new int[rsize];
	temp2 = new int[rsize];

	int j = threadIdx.x % (rsize*nd);
	int k = threadIdx.x / (rsize*nd);

	for (int i = k; i < (rsize*nd); i += stride)
	{
		i1 = i*rsize;
		i2 = (i + 1)*rsize - 1;
		j1 = j*rsize;
		j2 = (j + 1)*rsize - 1;
		maccess(d_M1, D, i1, i2, j1, j2, rsize*nd);
		msave(D, d_bigM, i1, i2, j1, j2, 0, 8, rsize*nd);
		rotmat(D, temp, rsize);
		msave(temp, d_bigM, i1, i2, j1, j2, 1, 8, rsize*nd);
		rotmat(temp, temp2, rsize);
		msave(temp2, d_bigM, i1, i2, j1, j2, 2, 8, rsize*nd);
		rotmat(temp2, temp, rsize);
		msave(temp, d_bigM, i1, i2, j1, j2, 3, 8, rsize*nd);
		fliph(D, temp, rsize);
		msave(temp, d_bigM, i1, i2, j1, j2, 4, 8, rsize*nd);
		flipv(D, temp, rsize);
		msave(temp, d_bigM, i1, i2, j1, j2, 5, 8, rsize*nd);
		transpose(D, temp, rsize);
		msave(temp, d_bigM, i1, i2, j1, j2, 6, 8, rsize*nd);
		rotmat(temp, temp2, rsize);
		rotmat(temp2, temp, rsize);
		msave(temp, d_bigM, i1, i2, j1, j2, 7, 8, rsize*nd);
	}

	delete[] temp;
	delete[] temp2;
	delete[] D;
}

__global__  void search(int *d_M, int *d_M1, int *T, int *d_bigM, int rsize, int nd, int nr, int sv, int sh)
{
	__shared__ int i1, i2, j1, j2, k1, k2, l1, l2, off, i0, j0, m0, s0, g0, del_g, sum_dist;
	__shared__ float dist, dmin;
	float s[4] = { 0.45, 0.60, 0.80, 1.00 };
	/*s[0] = 0.45;
	s[1] = 0.60;
	s[2] = 0.80;
	s[3] = 1.00;*/
	int domainDim = nd*nd*rsize*rsize * 8;
	__shared__ int *sharedBigM, *R, *D, *temp, *temp2;

	sharedBigM = new int[domainDim];
	R = new int[rsize*rsize];
	D = new int[rsize*rsize];
	temp = new int[rsize*rsize];
	temp2 = new int[rsize*rsize];

	for (int i = 0; i< domainDim; i++) 
		sharedBigM[i] = d_bigM[i];

	int k = blockIdx.x;
	int l = blockIdx.y;

	k1 = k*rsize;
	k2 = (k + 1)*rsize - 1;
	l1 = l*rsize;
	l2 = (l + 1)*rsize - 1;

	maccess(d_M, R, k1, k2, l1, l2, sv);
	off = mavg(R, rsize*rsize);

	dmin = (int)powf(10, 6);
	i0 = 0;
	j0 = 0;
	m0 = 0;

	for (int i = 0; i < nd; i++)
	{
		i1 = i*rsize;
		i2 = (i + 1)*rsize - 1;
		for (int j = 0; j < nd; j++)
		{
			j1 = j*rsize;
			j2 = (j + 1)*rsize - 1;
			// Test each transformation
			for (int n = 0; n < 4; n++)
			{
				for (int m = 0; m < 8; m++)
				{
					maccess(sharedBigM, D, i1, i2, j1, j2, m, 8, rsize*nd);
					scale(D, s[n], rsize*rsize);
					del_g = off - mavg(D, rsize*rsize);
					increment(D, del_g, rsize*rsize);
					diff(R, D, temp, rsize*rsize);
					msquare(temp, temp2, rsize*rsize);
					sum_dist = msum(temp2, rsize*rsize);
					dist = sqrtf(sum_dist);
					if (dist < dmin)
					{
						dmin = dist;
						i0 = i;
						j0 = j;
						m0 = m;
						s0 = n;
						g0 = del_g;
					}
				}
			}
		}
		T[((k*nr + l) * 5 + 0)] = i0;
		T[((k*nr + l) * 5 + 1)] = j0;
		T[((k*nr + l) * 5 + 2)] = m0;
		T[((k*nr + l) * 5 + 3)] = s0;
		T[((k*nr + l) * 5 + 4)] = g0;
	}
}

hipError_t cuda_encoder(int *h_r, int *h_g, int *h_b, int *h_Tr, int *h_Tg, int *h_Tb, int rsize, int nd, int nr, int sv, int sh)
{

	int *d_r, *d_g, *d_b, *d_Tr, *d_Tg, *d_Tb, *d_M1_r, *d_M1_g, *d_M1_b, *d_bigM_r, *d_bigM_g, *d_bigM_b;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&d_r, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_g, sv * sh*sizeof(int));
	cudaStatus = hipMalloc((void**)&d_b, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_Tr, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_Tg, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_Tb, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_M1_r, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_M1_g, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_M1_b, sv * sh* sizeof(int));
	cudaStatus = hipMalloc((void**)&d_bigM_r, nd*nd*rsize*rsize * 8 * sizeof(int));
	cudaStatus = hipMalloc((void**)&d_bigM_g, nd*nd*rsize*rsize * 8 * sizeof(int));
	cudaStatus = hipMalloc((void**)&d_bigM_b, nd*nd*rsize*rsize * 8 * sizeof(int));

	cudaStatus = hipMemcpy(d_r, h_r, sv * sh* sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_g, h_g, sv * sh* sizeof(int), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_b, h_b, sv * sh* sizeof(int), hipMemcpyHostToDevice);

	//kernel call here
	dim3 grid1 = (64, 1, 1);
	dim3 block1 = (1024, 1, 1);

	dim3 grid2 = (sv / rsize, sh / rsize, 1);
	dim3 block2 = (1, 1, 1);

	cuda_M1 <<<grid1,block1>>> (d_r, d_M1_r, rsize, nd, sv);

	cudaStatus = hipMemcpy(h_Tr, d_Tr, nr * nr * 5 * sizeof(int), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(h_Tg, d_Tg, nr * nr * 5 * sizeof(int), hipMemcpyDeviceToHost);
	cudaStatus = hipMemcpy(h_Tb, d_Tb, nr * nr * 5 * sizeof(int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	return cudaStatus;
}